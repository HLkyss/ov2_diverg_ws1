#include "hip/hip_runtime.h"
// virtual_image_kernel.cu
#include "/home/hl/project/ov2_diverg_ws/src/ov2slam/include/virtual_image_kernel.cuh"
#include <hip/hip_runtime.h>

// CUDA 核函数
__global__ void generateVirtualImageKernel(const unsigned char* realImage, unsigned char* virtualImage,
                                           int width, int height, int realWidth, int realHeight,
                                           const double* R, const double* virtualCameraK,
                                           const double* realCameraK, const double* T) {
    int v_x = blockIdx.x * blockDim.x + threadIdx.x;
    int v_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (v_x < width && v_y < height) {
        // 虚拟图像坐标归一化
        double point3D_virtual[3];
        point3D_virtual[0] = (v_x - virtualCameraK[2]) / virtualCameraK[0];
        point3D_virtual[1] = (v_y - virtualCameraK[5]) / virtualCameraK[4];
        point3D_virtual[2] = 1.0;

        // 将虚拟坐标转换到真实相机坐标
        double point3D_real[3];
        for (int i = 0; i < 3; i++) {
//            point3D_real[i] = R[0 * 3 + i] * point3D_virtual[0] +
//                              R[1 * 3 + i] * point3D_virtual[1] +
//                              R[2 * 3 + i] * point3D_virtual[2] - T[i];//使用转置的R矩阵
            point3D_real[i] = R[i * 3 + 0] * point3D_virtual[0] +
                              R[i * 3 + 1] * point3D_virtual[1] +
                              R[i * 3 + 2] * point3D_virtual[2] - T[i];//使用R
        }

        if (point3D_real[2] != 0) {
            // 将三维点投影到真实相机的像素坐标
            double realPixelX = (realCameraK[0] * point3D_real[0] + realCameraK[1] * point3D_real[1] + realCameraK[2] * point3D_real[2]) / point3D_real[2];
            double realPixelY = (realCameraK[3] * point3D_real[0] + realCameraK[4] * point3D_real[1] + realCameraK[5] * point3D_real[2]) / point3D_real[2];

            int pixelX = static_cast<int>(realPixelX);
            int pixelY = static_cast<int>(realPixelY);

            if (pixelX >= 0 && pixelX < realWidth && pixelY >= 0 && pixelY < realHeight) {
                virtualImage[v_y * width + v_x] = realImage[pixelY * realWidth + pixelX];
            } else {
                virtualImage[v_y * width + v_x] = 0;
            }
        } else {
            virtualImage[v_y * width + v_x] = 0;
        }
    }
}

// CUDA 接口函数
void generateVirtualImageCUDA(const unsigned char* realImage, unsigned char* virtualImage, int width, int height,
                              int realWidth, int realHeight, const double* R, const double* virtualCameraK,
                              const double* realCameraK, const double* T) {
    // 分配并复制矩阵数据到设备
    double *d_R, *d_virtualCameraK, *d_realCameraK, *d_T;
    hipMalloc((void**)&d_R, 9 * sizeof(double));
    hipMalloc((void**)&d_virtualCameraK, 9 * sizeof(double));
    hipMalloc((void**)&d_realCameraK, 9 * sizeof(double));
    hipMalloc((void**)&d_T, 3 * sizeof(double));

    hipMemcpy(d_R, R, 9 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_virtualCameraK, virtualCameraK, 9 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_realCameraK, realCameraK, 9 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_T, T, 3 * sizeof(double), hipMemcpyHostToDevice);

    // 分配并复制图像数据到设备
    unsigned char *d_realImage, *d_virtualImage;
    hipMalloc(&d_realImage, realWidth * realHeight * sizeof(unsigned char));
    hipMalloc(&d_virtualImage, width * height * sizeof(unsigned char));
    hipMemcpy(d_realImage, realImage, realWidth * realHeight * sizeof(unsigned char), hipMemcpyHostToDevice);

    // 配置 CUDA 网格和块大小
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // 执行 CUDA 内核
    generateVirtualImageKernel<<<gridSize, blockSize>>>(d_realImage, d_virtualImage, width, height, realWidth, realHeight, d_R, d_virtualCameraK, d_realCameraK, d_T);

    // 复制结果回主机
    hipMemcpy(virtualImage, d_virtualImage, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    // 释放设备内存
    hipFree(d_R);
    hipFree(d_virtualCameraK);
    hipFree(d_realCameraK);
    hipFree(d_T);
    hipFree(d_realImage);
    hipFree(d_virtualImage);
}
